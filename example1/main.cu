/**
 * Simple CUDA application template.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN( value ) {							\
	hipError_t err = value;									\
	if( err != hipSuccess ) {									\
		fprintf( stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(err), __LINE__, __FILE__ );	\
		exit( 1 );												\
	} }

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char **argv) {
	std::cout << "Hello in CUDA World!" << std::endl;
	
	int devices = 0;
	
	CUDA_CHECK_RETURN( hipGetDeviceCount( &devices ) );
	
	std::cout << "Available devices: " << devices << std::endl;
	
	hipDeviceProp_t properties;
	
	for( int i = 0; i < devices; ++i ) {
		CUDA_CHECK_RETURN( hipGetDeviceProperties( &properties, i ) );
		std::cout << "Device " << i << " name: " << properties.name << std::endl;
		std::cout << "Compute capability: " << properties.major << "." << properties.minor << std::endl;
		std::cout << "Block dimensions: " << properties.maxThreadsDim[0]
			<< ", " << properties.maxThreadsDim[1]
			<< ", "<< properties.maxThreadsDim[2]
			<< std::endl;
		std::cout << "Grid dimensions: " << properties.maxGridSize[0]
			<< ", " << properties.maxGridSize[1]
			<< ", " << properties.maxGridSize[2]
			<< std::endl;
	}
	
	return 0;
}
